#include "hip/hip_runtime.h"
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

// Run in a single block
// Calculate bilayer COM
extern "C" __global__ void computeOrigin( 
const real4* __restrict__ posq,
double* origin,
double* origin_buffer,
const int* __restrict__ particles_for_origin, 
const double* __restrict__ mass_for_origin
) {
    int threadIndex = threadIdx.x;

    origin_buffer[threadIndex] = 0.0;

    // Calculate COM
    for (int index=threadIndex; index<ORIGIN_NUM_ATOMS; index+=blockDim.x) {
        origin_buffer[threadIndex] += mass_for_origin[index] * posq[particles_for_origin[index]].z;
    }
    __syncthreads();

    // Do parallel reduction for origin_buffer
    for (unsigned int stride=blockDim.x/2; stride>0; stride>>=1) {
        if (threadIndex < stride) {
            origin_buffer[threadIndex] += origin_buffer[threadIndex + stride];
        }
        __syncthreads();
    }
    __syncthreads();
    
    if (threadIndex == 0) {
        origin[0] = origin_buffer[0] / total_mass_origin;
    }
}

// Run in a single block
extern "C" __global__ void computePreFFtotal(
double* A_real_xray_current, 
double* A_real_neutron_current, 
double* A_complex_xray_current, 
double* A_complex_neutron_current
) {
    int threadIndex = threadIdx.x;

    // Init A_current
    // Set the A to 0.0 to avoid porblems on reduction step if blockDim.x > Nq_xray or Nq_neutron
    for (int i=0; i<Nq_xray; ++i){
        A_real_xray_current[i*blockDim.x + threadIndex] = 0.0;
        A_complex_xray_current[i*blockDim.x + threadIndex] = 0.0;
    }
    for (int i=0; i<Nq_neutron; ++i){
        A_real_neutron_current[i*blockDim.x + threadIndex] = 0.0;
        A_complex_neutron_current[i*blockDim.x + threadIndex] = 0.0;
    }
}

// Calculate form factor components for the current system
extern "C" __global__ void computeFFtotal( 
const real4* __restrict__ posq,
double* origin,
const float* __restrict__ box, 
const int* __restrict__ is_water, 
const float* __restrict__ xray_strength, 
const float* __restrict__ neutron_strength, 
const char* __restrict__ atom_names,
const float* __restrict__ xray_qs, 
const float* __restrict__ neutron_qs, 
const float* __restrict__ cutoff, 
const float* __restrict__ d_parts, 
double* A_real_xray_current, 
double* A_real_neutron_current, 
double* A_complex_xray_current, 
double* A_complex_neutron_current
) {
    int threadIndex = threadIdx.x;

    // Now the system itself. A_real_current, A_complex_current
    for (int index=blockIdx.x * blockDim.x + threadIndex; index<Natoms; index+=blockDim.x * gridDim.x) {

        // Wrap the coords of the atom
        float z = posq[index].z - origin[0];
	if (fabsf(z) > box[2]/2.0) {
            if (z<0.0) {
                z = z - box[2]*floorf((z - 0.5*(box[2]))/box[2]);
            }
            if (z>=0.0) {
                z = z - box[2]*floorf((z + 0.5*(box[2]))/box[2]);
            }
        }

	if (fabsf(z)<=cutoff[0]){
    	    // X-ray
	    for (int i=0; i<Nq_xray; ++i){
		atomicAdd(&A_real_xray_current[i*blockDim.x + threadIndex], xray_strength[index*Nq_xray + i] * cosf(xray_qs[i]*z));
		atomicAdd(&A_complex_xray_current[i*blockDim.x + threadIndex], xray_strength[index*Nq_xray + i] * sinf(xray_qs[i]*z));
    	    }
    	    // Neutron
    	    for (int i=0; i<Nq_neutron; ++i){
		atomicAdd(&A_real_neutron_current[i*blockDim.x + threadIndex], neutron_strength[index*Nq_neutron + i] * cosf(neutron_qs[i]*z));
    	    	atomicAdd(&A_complex_neutron_current[i*blockDim.x + threadIndex], neutron_strength[index*Nq_neutron + i] * sinf(neutron_qs[i]*z));
    	    }
	}
    }
}

// Run in a single block
extern "C" __global__ void computePostFFtotal( 
double* A_real_xray_current, 
double* A_real_neutron_current, 
double* A_complex_xray_current, 
double* A_complex_neutron_current, 
double* A_real_xray_out, 
double* A_real_neutron_out, 
double* A_complex_xray_out, 
double* A_complex_neutron_out, 
double* A_sqr_xray_out,
double* A_sqr_neutron_out
) {
    int threadIndex = threadIdx.x;

    // Reduce to have one value for each q and calculate forces

    // Reduce X-ray
    for (int i=0; i<Nq_xray; ++i){
        for (unsigned int stride=blockDim.x/2; stride>0; stride>>=1) {
            if (threadIndex < stride) {
                A_real_xray_current[threadIndex + i*blockDim.x] += A_real_xray_current[threadIndex + i*blockDim.x + stride];
                A_complex_xray_current[threadIndex + i*blockDim.x] += A_complex_xray_current[threadIndex + i*blockDim.x + stride];
            }
            __syncthreads();
        }
    }
    __syncthreads();

    // We have the reduced values in A_real_xray_current[i*blockDim.x] for each i-th q value
    // Now put them into the first Nq_xray elements of A_real_xray_current for a future use
    for (int index=threadIndex; index<Nq_xray; index+=blockDim.x) {
	A_real_xray_out[index] = A_real_xray_current[index*blockDim.x];
	A_complex_xray_out[index] = A_complex_xray_current[index*blockDim.x];
	A_sqr_xray_out[index] = A_real_xray_current[index*blockDim.x]*A_real_xray_current[index*blockDim.x] + A_complex_xray_current[index*blockDim.x]*A_complex_xray_current[index*blockDim.x];
    }

    // Reduce neutron
    for (int i=0; i<Nq_neutron; ++i){
	for (unsigned int stride=blockDim.x/2; stride>0; stride>>=1) {
            if (threadIndex < stride) {
                A_real_neutron_current[threadIndex + i*blockDim.x] += A_real_neutron_current[threadIndex + i*blockDim.x + stride];
                A_complex_neutron_current[threadIndex + i*blockDim.x] += A_complex_neutron_current[threadIndex + i*blockDim.x + stride];
            }
            __syncthreads();
        }
    }
    __syncthreads();
    
    // We have the reduced values in A_real_neutron_current[i*blockDim.x] for each i-th q value
    // Now put them into the first Nq_neutrin elements of A_real_neutron_current for a future use
    for (int index=threadIndex; index<Nq_neutron; index+=blockDim.x) {
	A_real_neutron_out[index] = A_real_neutron_current[index*blockDim.x];
	A_complex_neutron_out[index] = A_complex_neutron_current[index*blockDim.x];
	A_sqr_neutron_out[index] = A_real_neutron_current[index*blockDim.x]*A_real_neutron_current[index*blockDim.x] + A_complex_neutron_current[index*blockDim.x]*A_complex_neutron_current[index*blockDim.x];
    }
}

// Here we precalculate B_real/B_sqr
extern "C" __global__ void computePreForce( 
const real4* __restrict__ posq,
const float* __restrict__ box, 
const float* __restrict__ xray_h_strength, 
const float* __restrict__ xray_o_strength, 
const float* __restrict__ xray_qs, 
const float* __restrict__ neutron_qs, 
const float* __restrict__ neutron_o_strength, 
const float* __restrict__ neutron_h_strength, 
const float* __restrict__ neutron_d_strength, 
const double* __restrict__ w_dens, 
const double* __restrict__ w_dens_sqr, 
const float* __restrict__ cutoff, 
const float* __restrict__ d_parts, 
double* B_real_xray_current, 
double* B_real_neutron_current,
double* B_sqr_xray_current, 
double* B_sqr_neutron_current 
) {
    int threadIndex = threadIdx.x;

    // Get B_real
    // Xray
    for (int index=threadIndex; index<Nq_xray; index+=blockDim.x) {
        const float wXrayStrength = 2.0 * xray_h_strength[index] * (1.0 + (-0.48)*exp(-xray_qs[index]*xray_qs[index]/(2*0.22*0.22))) +
                                          xray_o_strength[index] * (1.0 + 0.12*exp(-xray_qs[index]*xray_qs[index]/(2*0.22*0.22)));
        B_real_xray_current[index] = 2.0 * w_dens[0] * box[0] * box[1] * wXrayStrength * sinf(xray_qs[index]*cutoff[0]) / xray_qs[index];
	B_sqr_xray_current[index] = w_dens_sqr[0] * (2.0*box[0] * box[1] * wXrayStrength * sinf(xray_qs[index]*cutoff[0]) / xray_qs[index]) 
					  * (2.0*box[0] * box[1] * wXrayStrength * sinf(xray_qs[index]*cutoff[0]) / xray_qs[index]);
    }
    
    // Neutron
    for (int index=threadIndex; index<Nq_neutron; index+=blockDim.x) {
        const double w_neutr_scatt_streng = neutron_o_strength[0] + 2.0*(d_parts[index] * neutron_d_strength[0] + (1.0 - d_parts[index]) * neutron_h_strength[0]);
        B_real_neutron_current[index] = 2.0 * w_dens[0] * box[0] * box[1] * w_neutr_scatt_streng * sinf(neutron_qs[index]*cutoff[0]) / neutron_qs[index];
	B_sqr_neutron_current[index] = w_dens_sqr[0] * (2.0*box[0] * box[1] * w_neutr_scatt_streng * sinf(neutron_qs[index]*cutoff[0]) / neutron_qs[index]) 
					     * (2.0*box[0] * box[1] * w_neutr_scatt_streng * sinf(neutron_qs[index]*cutoff[0]) / neutron_qs[index]);
    }
}

// Calculate force
extern "C" __global__ void computeForce( 
const real4* __restrict__ posq,
const float* __restrict__ alpha, 
double* origin,
const double* __restrict__ k_xray, 
const double* __restrict__ k_neutron, 
const double* __restrict__ T, 
const float* __restrict__ box, 
const int* __restrict__ is_water, 
const float* __restrict__ xray_strength, 
const float* __restrict__ neutron_strength, 
const char* __restrict__ atom_names,
const float* __restrict__ xray_qs, 
const float* __restrict__ neutron_qs, 
const float* __restrict__ cutoff, 
const float* __restrict__ d_parts, 
double* B_real_xray_current, 
double* B_real_neutron_current,
double* A_real_xray_out, 
double* A_real_neutron_out, 
double* A_complex_xray_out, 
double* A_complex_neutron_out, 
double* F_total_xray, 
double* F_total_neutron, 
const int* __restrict__ particles, 
const double* __restrict__ F_exp_xray,
const double* __restrict__ F_exp_neutron,
const double* __restrict__ delta_F_exp_xray,
const double* __restrict__ delta_F_exp_neutron,
unsigned long long* __restrict__ forceBuffer
) {
    int threadIndex = threadIdx.x;

    const double const_xray = (-2.0) * alpha[0] * k_xray[0]* T[0] * (1.380658e-23 * 6.0221367e23 /1e3) * (1.0/Nq_xray); // k_Boltzmann (J*K^-1) * N_avagadro (mol^-1) * 10e-3 (J->kJ) = 0.0083144621 (kJ/(mol*K))
    const double const_neutron = (-2.0) * alpha[0] * k_neutron[0]* T[0] * (1.380658e-23 * 6.0221367e23 /1e3) * (1.0/Nq_neutron); // k_Boltzmann (J*K^-1) * N_avagadro (mol^-1) * 10e-3 (J->kJ) = 0.0083144621 (kJ/(mol*K))
    for (int index=blockIdx.x * blockDim.x + threadIndex; index<particles_size; index+=blockDim.x * gridDim.x) {
        // Wrap the coords of the atom
        float z = posq[particles[index]].z - origin[0];
        if (fabsf(z) > box[2]/2.0) {
            if (z<0.0) {
                z = z - box[2]*floorf((z - 0.5*(box[2]))/box[2]);
            }
            if (z>=0.0) {
                z = z - box[2]*floorf((z + 0.5*(box[2]))/box[2]);
            }
        }
	if (fabsf(z)<=cutoff[0]){ 
    	    double scatt_streng_for_force;
	    // Collect force for the atom number particles[index]
	    double force = 0.0;

    	    // X-ray
	    for (int i=0; i<Nq_xray; ++i){
        	scatt_streng_for_force = xray_strength[particles[index]*Nq_xray + i];
        	force += const_xray * ((F_total_xray[i] - F_exp_xray[i]) / (delta_F_exp_xray[i]*delta_F_exp_xray[i])) *
                            ((1.0/F_total_xray[i]) * scatt_streng_for_force * xray_qs[i] * 
			     (-sinf(xray_qs[i]*z)*(A_real_xray_out[i] - B_real_xray_current[i]) + cosf(xray_qs[i]*z)*A_complex_xray_out[i]));
	    }
    	    // Neutron
    	    for (int i=0; i<Nq_neutron; ++i){
        	scatt_streng_for_force = neutron_strength[particles[index]*Nq_neutron + i];
		force += const_neutron * ((F_total_neutron[i] - F_exp_neutron[(i)]) / (delta_F_exp_neutron[i]*delta_F_exp_neutron[i])) *
                            ((1.0/F_total_neutron[i]) * scatt_streng_for_force * neutron_qs[i] * 
			     (-sinf(neutron_qs[i]*z)*(A_real_neutron_out[i] - B_real_neutron_current[i]) + cosf(neutron_qs[i]*z)*A_complex_neutron_out[i]));
    	    }
	
	    // Add the result
	    atomicAdd(&forceBuffer[particles[index]+2*PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long)(force*0x100000000)));
	}
    }
}

// Calculate energy
extern "C" __global__ void computeEnergy( 
const float* __restrict__ alpha, 
float* energy_buffer,
const double* __restrict__ k_xray, 
const double* __restrict__ k_neutron, 
const double* __restrict__ T, 
double* F_total_xray, 
double* F_total_neutron, 
const double* __restrict__ F_exp_xray,
const double* __restrict__ F_exp_neutron,
const double* __restrict__ delta_F_exp_xray,
const double* __restrict__ delta_F_exp_neutron,
real* __restrict__ energyBuffer
) {
    int threadIndex = threadIdx.x;
    
    // Compute energy

    // Zero out
    energy_buffer[threadIndex] = 0.0;

    for (int index=threadIndex; index<Nq_xray; index+=blockDim.x) {// k_B (J*K^-1) * N_avagadro (mol^-1) * 10e-3 (J->kJ) = 0.0083144621 (kJ/(mol*K))
        energy_buffer[threadIndex] += alpha[0] * k_xray[0]* T[0] * (1.380658e-23 * 6.0221367e23 /1e3) * (1.0/Nq_xray) * ((F_total_xray[index] - F_exp_xray[index])*(F_total_xray[index] - F_exp_xray[index]) / (delta_F_exp_xray[index]*delta_F_exp_xray[index]));
    }
    __syncthreads();

    for (int index=threadIndex; index<Nq_neutron; index+=blockDim.x) {// k_B (J*K^-1) * N_avagadro (mol^-1) * 10e-3 (J->kJ) = 0.0083144621 (kJ/(mol*K))
        energy_buffer[threadIndex] += alpha[0] * k_neutron[0]* T[0] * (1.380658e-23 * 6.0221367e23 /1e3) * (1.0/Nq_neutron) * ((F_total_neutron[index] - F_exp_neutron[index])*(F_total_neutron[index] - F_exp_neutron[index]) / (delta_F_exp_neutron[index]*delta_F_exp_neutron[index]));
    }
    __syncthreads();

    // Reduce energy
    for (unsigned int stride=blockDim.x/2; stride>0; stride>>=1) {
        if (threadIndex < stride) {
            energy_buffer[threadIndex] += energy_buffer[threadIndex + stride];
        }
        __syncthreads();
    }

    if (threadIndex == 0) {
        energyBuffer[0] += energy_buffer[0];
    }
}

// This kernel is used if we set on_gpu flag to true for updateParametersInContext(context, on_gpu)
// If on_gpu is set to false we do the form factor calculation on the host
extern "C" __global__ void computeGlobalFtotal( 
const double* __restrict__ A_real_xray_out,
const double* __restrict__ A_complex_xray_out,
const double* __restrict__ A_sqr_xray_out,
const double* __restrict__ A_real_neutron_out,
const double* __restrict__ A_complex_neutron_out,
const double* __restrict__ A_sqr_neutron_out,
const double* __restrict__ B_real_xray_global,
const double* __restrict__ B_sqr_xray_global,
const double* __restrict__ B_real_neutron_global,
const double* __restrict__ B_sqr_neutron_global,
double* F_total_xray,
double* F_total_neutron
) {
    int threadIndex = threadIdx.x;

    for (int index=blockIdx.x * blockDim.x + threadIndex; index<Nq_xray; index+=blockDim.x * gridDim.x) {
	double intens = A_real_xray_out[index]*A_real_xray_out[index] + A_complex_xray_out[index]*A_complex_xray_out[index]
                                      + B_real_xray_global[index]*B_real_xray_global[index] - 2.0*A_real_xray_out[index]*B_real_xray_global[index]
                                      + A_sqr_xray_out[index] - A_real_xray_out[index]*A_real_xray_out[index] - A_complex_xray_out[index]*A_complex_xray_out[index]
                                      - B_sqr_xray_global[index] + B_real_xray_global[index]*B_real_xray_global[index];
	F_total_xray[index] = copysignf(1.0, intens)*sqrt(fabs(intens));
    }

    for (int index=blockIdx.x * blockDim.x + threadIndex; index<Nq_neutron; index+=blockDim.x * gridDim.x) {
	double intens = A_real_neutron_out[index]*A_real_neutron_out[index] + A_complex_neutron_out[index]*A_complex_neutron_out[index]
                                      + B_real_neutron_global[index]*B_real_neutron_global[index] - 2.0*A_real_neutron_out[index]*B_real_neutron_global[index]
                                      + A_sqr_neutron_out[index] - A_real_neutron_out[index]*A_real_neutron_out[index] - A_complex_neutron_out[index]*A_complex_neutron_out[index]
                                      - B_sqr_neutron_global[index] + B_real_neutron_global[index]*B_real_neutron_global[index];
	F_total_neutron[index] = copysignf(1.0, intens)*sqrt(fabs(intens));
    }
    __syncthreads();
}
